#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

extern "C"
__global__ void sde_curand_kernel(
    float* __restrict__ paths,
    int    n_steps,
    int    stride,
    float  T,
    float  feedback_value,
    float  decay_rate,
    float  high_threshold,
    float  low_threshold,
    float  total_steps,
    float  base_variance
) {
    int pid = blockIdx.x * blockDim.x + threadIdx.x;
    float dt = T / n_steps;
    // initialize RNG per-thread
    hiprandStatePhilox4_32_10_t state;
    hiprand_init( /* seed */ 1234ULL, /* subsequence */ pid, /* offset */ 0, &state);
    // load initial
    float acc = paths[pid * stride];
    for (int i = 0; i < n_steps; ++i) {
        // draw a Gaussian
        float rnd = hiprand_normal(&state);

        // control_randomness_by_state
        float norm = 1.0f / total_steps;
        float factor1 = (norm < low_threshold ? 1.5f
                          : (norm > high_threshold ? 0.5f : 1.0f));
        float t  = i * dt;
        float vf = base_variance * factor1 * expf(-decay_rate * t);
        float scale2 = fminf(1.0f, fmaxf(0.2f, 0.5f + feedback_value * 0.5f));

        // accumulate
        float inc = rnd * sqrtf(dt) * scale2 * vf;
        acc += inc;
        paths[pid * stride + i + 1] = acc;
    }
}
